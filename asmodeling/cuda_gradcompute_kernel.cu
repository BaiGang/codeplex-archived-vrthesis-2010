#ifndef __CUDA_GRAD_COMPUTE_KERNEL_CU__
#define __CUDA_GRAD_COMPUTE_KERNEL_CU__

#include <hip/hip_runtime.h>

// Kernel function to calculate the sum-of-square-error
__global__ void calc_f(float * ground_truth, float * render_result, int n, float * f_pixels)
{
  //  calc index of the thread
  int thread_ind = threadIdx.y * blockDim.x + threadIdx.x;

  //  calc f for each pixel
  f_pixels[ thread_ind ] += (ground_truth[thread_ind] - render_result[thread_ind])
    * (ground_truth[thread_ind] - render_result[thread_ind]);

  __syncthreads();

  // reduce and sum
  //  the final result will be stored in f_pixels[0]
  for (int i = n; i > 0; i /= 2)
  {
    if (thread_ind < i)
    {
      f_pixels[thread_ind] = f_pixels[thread_ind] + f_pixels[thread_ind +i];
    }

    __syncthreads();
  }
}

// Kernel function to calculate the gradient
__global__ void calc_g(float * ground_truth,
                       float * render_res_imperturbed,
                       float * render_res_perturbed,
                       int *   index_pixels2arrayg,
                       float * o_g,
                       int n,
                       int n_pixels)
{

}


////////////////////////////////////////////////////////////////
//
//   Perturb the voxels to calc gradient
//   The serial edition of the algorithm
//
//First, calc the most close axis to the view direction. This axis
//is the principal axis.
//
//N = the resolution of the slice
//M = number of sub slices
//
//For each slice along the principal axis
//  for group_v = 0 to N/M-1
//    for group_u = 0 to N/M-1  // for each concurrently perturbed group
//    {
//      // firstly, perturb
//
//      for sub_v = 0 to M
//        for sub_u = 0 to M    // for each sub slice
//        {
//          if this voxel contains density
//            perturb it
//        }
//
//      // then, render
//      Render 
//
//      // Sum the gradient
//
//    }


#endif //__CUDA_GRAD_COMPUTE_KERNEL_CU__