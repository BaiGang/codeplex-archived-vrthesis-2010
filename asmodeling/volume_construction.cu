#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////////////////////////
//
//  Reconstruct the volume on the cuda
//
/////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void construct_volume(
                                 hipPitchedPtr vol_pptr, 
                                 float * device_x, 
                                 int * tag_vol
                                 )
{
  //unsigned int z = blockIdx.x;
  //unsigned int y = blockIdx.y;
  //unsigned int x = threadIdx.x;
  // slice_pitch = pitch * height
  // slice = ptr + z * slice_pitch
  // row = (type*)(slice + y*pitch)
  // elem = *(row + x)

  int index = index3(threadIdx.x, blockIdx.y, blockIdx.x, blockDim.x);

  char * slice = (char *)vol_pptr.ptr + blockIdx.x * vol_pptr.pitch * blockDim.x;

  *((float*)(slice+blockIdx.y*vol_pptr.pitch) + threadIdx.x)
    = device_x[ tag_vol[index] ];
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//
//  Upsample current level to max_level
//
/////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void upsample_volume(
                                hipPitchedPtr pptr_lower,
                                hipExtent     extent_lower,
                                hipPitchedPtr pptr_higher,
                                int scale
                                )
{
  // for higher level volume
  unsigned int k = blockIdx.x;
  unsigned int j = blockIdx.y;
  unsigned int i = threadIdx.x;

  char * slice = (char*)pptr_higher.ptr + k*pptr_higher.pitch*blockDim.x;
  float *p_higher = (float*)(slice + j*pptr_higher.pitch) + i;

  k /= scale;
  j /= scale;
  k /= scale;

  slice = (char*)pptr_lower.ptr + k*pptr_lower.pitch*extent_lower.depth;
  float *p_lower  = (float*)(slice + j*pptr_lower.pitch) + i;

  *p_higher = *p_lower;
}