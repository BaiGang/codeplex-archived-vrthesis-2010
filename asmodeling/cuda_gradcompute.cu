#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <GL/glew.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>


#include "cuda_util.cuh"


/////////////////////////////////////////////////////////////
//
//       GLOBAL VARIABLES on both DEVICE and HOST
//      "static" to prevent solution on namespace
//
/////////////////////////////////////////////////////////////

// the level of volume for current optimization pass
static int current_level;

// the cuda 3D array for tag volume
static hipArray * tag_volume;
// the 3D texture reference for tag volume
static texture<int, 3, hipReadModeElementType> voltag_tex;

// the pitched pointer to the density 3D volume
static hipPitchedPtr density_volume_pptr;


/////////////////////////////////////////////////////////////
//
//       DEVICE CODE
//
/////////////////////////////////////////////////////////////

// construct the density volume
// the volume is a length^3 cube
// each z is dispatched to blocks
// x, y are within a block
__global__ void construct_density_volome(hipPitchedPtr vol_pptr, float * device_x, int length)
{
  unsigned int z = blockIdx.x;
  unsigned int y = threadIdx.y;
  unsigned int x = threadIdx.x;

  if(z >= length || y >= length || x >= length)
  {
    return;
  }

  int texfch = tex3D(voltag_tex, x, y, z);

  size_t slice_pitch = vol_pptr.pitch * length;

  *(float*)((char*)vol_pptr.ptr + z*slice_pitch + y * vol_pptr.pitch +x) = device_x[texfch];
}



/////////////////////////////////////////////////////////////
//
//       HOST CODE FOR GLOBAL INIT
//
/////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////
//
//       HOST CODE FOR CURRENT FRAME INIT
//
/////////////////////////////////////////////////////////////
// set the level and the volume tags
// so that we can construct the density volume
// on the GPU side
extern "C"
void set_density_indicator(int level, int * tags)
{
  current_level = level;
  int length = 1<<level;

  hipExtent extent = make_hipExtent(length, length, length);
  hipChannelFormatDesc channel_desc = {0};
  channel_desc.f = hipChannelFormatKindSigned;
  channel_desc.x = 32;
  channel_desc.y = 0;
  channel_desc.z = 0;
  channel_desc.w = 0;

  hipMalloc3DArray(&tag_volume, &channel_desc, extent);

  hipMemcpy3DParms copy_to_indicator = {0};
  copy_to_indicator.srcPtr = make_hipPitchedPtr((void*)tags, length*sizeof(int), length, length);
  copy_to_indicator.dstArray = tag_volume;
  copy_to_indicator.extent = extent;
  copy_to_indicator.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&copy_to_indicator);

  voltag_tex.normalized = false;
  voltag_tex.filterMode = hipFilterModePoint;
  voltag_tex.addressMode[0] = hipAddressModeWrap;
  voltag_tex.addressMode[1] = hipAddressModeWrap;
  voltag_tex.addressMode[2] = hipAddressModeWrap;

  hipBindTextureToArray(&voltag_tex, tag_volume, &channel_desc);

  // mem aloc for density volume
  hipMalloc3D(&density_volume_pptr, extent);

  // get last error
}

extern "C"
void delete_density_indicator()
{
  hipFreeArray( tag_volume );
  hipFree( &density_volume_pptr );
}

/////////////////////////////////////////////////////////////
//
//       HOST CODE FOR CURRENT LEVEL INIT
//
/////////////////////////////////////////////////////////////

// construct the volume on GPU side using
// the array x and volume indicator
extern "C"
void construct_volume_cuda(float * p_x)
{
  int length = 1<<current_level;

  dim3 grid_dim(length, 1, 1);
  dim3 block_dim(length, length, 1);

  // run the kernel to construct volume on device


}

extern "C"
void subdivide_volume_cuda(int prev_level, int next_level)
{

}

// the main routine for grad computation
extern "C"
float cuda_grad_compute(float * p_host_x, float * p_host_g, int n)
{
  float * p_device_x ;
  float * p_device_g ;

  size_t size = n * sizeof(float);

  hipMalloc((void**)p_device_g, size);
  hipMalloc((void**)p_device_x, size+1);
  hipMemcpy(p_device_x, p_host_x, size+1, hipMemcpyHostToDevice);

  // construct volume
  construct_volume_cuda( p_device_x );

  // render to image 1


  // calc f

  // perturb voxel

  // render to image 2

  // calc g[]

  // copy g[] from device to host 
  hipMemcpy(p_host_g, p_device_g, size, hipMemcpyDeviceToHost);

  hipFree(p_device_x);
  hipFree(p_device_g);

  return 0.0f;
}

