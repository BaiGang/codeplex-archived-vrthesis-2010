#include <hip/hip_runtime.h>



extern "C" // functions of cuda-raymarching
{

}

// Kernel function to calculate the sum-of-square-error
__global__ void calc_f(float * ground_truth, float * render_result, int n, float * f_pixels)
{
  ////  calc index of the thread
  //int thread_ind = threadIdx.y * blockDim.x + threadIdx.x;

  ////  calc f for each pixel
  //f_pixels[ thread_ind ] += (ground_truth[thread_ind] - render_result[thread_ind])
  //  * (ground_truth[thread_ind] - render_result[thread_ind]);

  //__syncthreads();

  //// reduce and sum
  ////  the final result will be stored in f_pixels[0]
  //for (int i = n; i > 0; i /= 2)
  //{
  //  if (thread_ind < i)
  //  {
  //    f_pixels[thread_ind] = f_pixels[thread_ind] + f_pixels[thread_ind +i];
  //  }

  //  __syncthreads();
  //}
}

// Kernel function to calculate the gradient
__global__ void calc_g(float * ground_truth,
                       float * render_res_imperturbed,
                       float * render_res_perturbed,
                       int *   index_pixels2arrayg,
                       float * o_g,
                       int n,
                       int n_pixels)
{

}

// perturb voxels
__global__ void perturb_voxels()
{

}


////////////////////////////////////////////////////////////////
//
//   Perturb the voxels to calc gradient
//   The serial edition of the algorithm
//
//First, calc the most close axis to the view direction. This axis
//is the principal axis.
//
//N = the resolution of the slice
//M = number of sub slices
//
//For each slice along the principal axis
//  for group_v = 0 to N/M-1
//    for group_u = 0 to N/M-1  // for each concurrently perturbed group
//    {
//      // firstly, perturb
//
//      for sub_v = 0 to M
//        for sub_u = 0 to M    // for each sub slice
//        {
//          if this voxel contains density
//            perturb it
//        }
//
//      // then, render
//      Render 
//
//      // Sum the gradient
//
//    }

extern "C"
float cuda_grad_compute(float * p_host_x, float * p_host_g, int n)
{
  return 0.0f;
}
