
#include <cstdio>

#include <hip/hip_runtime.h>
#include <cutil_inline.h>
#include <cutil_math.h>

#include "devfun_def.h"

#include "utils.cuh"
#include "volume_construction.cu"
#include "f_calculation.cu"
#include "g_calculation.cu"

typedef unsigned short uint16;

void construct_volume_cuda (float * device_x,
                            hipPitchedPtr * density_vol,
                            hipExtent extent,
                            int *   tag_vol )
{
  dim3 grid_dim(extent.depth, extent.depth, 1);
  dim3 block_dim(extent.depth, 1, 1);

  fprintf(stderr, "Lauching Kernel \"construct_volume\", <<<(%d %d %d),(%d %d %d)>>>",
    extent.depth, extent.depth, 1, extent.depth, 1, 1);

  construct_volume<<< grid_dim, block_dim >>>(
    *density_vol,
    device_x,
    tag_vol
    );

  // check if kernel execution generated an error
  cutilCheckMsg("Kernel execution failed");
}

void construct_volume_linm_cuda (int length,
                                 float *devic_x,
                                 float *density_vol,
                                 int * tag_vol )
{
  dim3 grid_dim(length, length, 1);
  dim3 block_dim(length, 1, 1);

  construct_volume_linm<<<grid_dim, block_dim>>>(
    density_vol,
    devic_x,
    tag_vol );

  // check if kernel execution generated an error
  cutilCheckMsg("Kernel execution failed");

}

void upsample_volume_cuda (int level,
                           int max_level,
                           hipPitchedPtr * lower_lev,
                           hipPitchedPtr * upper_lev
                           )
{
  int length = 1 << level;
  int max_length = 1 << max_level;

  dim3 grid_dim(max_length, max_length, 1);
  dim3 block_dim(max_length, 1, 1);

  hipExtent ext_low = make_hipExtent(length, length, length);
  int scale = max_length / length;

  fprintf(stderr, "<+> Lauching kernel \"upsampe_volume\",upper: %d, lower: %d, scale: %d\n",
    max_length, length, scale);

  upsample_volume<<< grid_dim, block_dim >>>(
    *lower_lev,
    ext_low,
    *upper_lev,
    scale );

  // check if kernel execution generated an error
  cutilCheckMsg("Kernel execution failed");
}

void construct_volume_from_previous_cuda (
  float * device_x,
  hipPitchedPtr * density_vol,
  hipExtent extent,
  int * tag_vol
  )
{
  dim3 grid_dim(extent.depth, extent.depth, 1);
  dim3 block_dim(extent.depth, 1, 1);

  construct_volume_from_prev<<<grid_dim, block_dim>>> (
    *density_vol,
    device_x,
    tag_vol
    );

  cutilCheckMsg("Kernel execution failed");
}

void cull_empty_cells_cuda (hipPitchedPtr * density_vol,
                            hipExtent extent,
                            int * tag_vol )
{
  dim3 grid_dim(extent.depth, extent.depth, 1);
  dim3 block_dim(extent.depth, 1 ,1);

  cull_empty_cells<<<grid_dim, block_dim>>> (
    *density_vol,
    tag_vol );

  cutilCheckMsg("Kernel execution failed");
}

void get_guess_x_cuda (float * guess_x,
                       hipPitchedPtr * density_vol,
                       hipExtent extent,
                       int * tag_vol )
{
  dim3 grid_dim(extent.depth, extent.depth, 1);
  dim3 block_dim(extent.depth, 1, 1);

  get_guess_x<<<grid_dim, block_dim>>> (
    *density_vol,
    tag_vol,
    guess_x );

  cutilCheckMsg("Kernel execution failed");
}

extern "C"
void reduceSinglePass (int size,
                       int threads,
                       int blocks,
                       float *d_idata,
                       float *d_odata );

extern "C"
void reduce (int size,
             int threads,
             int blocks,
             float *d_idata,
             float *d_odata );

float calculate_f_cuda (int      level, 
                        int      i_view, 
                        int      n_view,
                        int      n_nonzero_items,
                        int      powtwo_length,
                        int      interval,
                        uint16*  projected_centers, 
                        int*     vol_tag,
                        float*   f_array,
                        float*   sum_array )
{
  int size = 1 << level;

  // calc f value for each non-zero cell
  dim3 grid_dim(size, size, 1);
  dim3 block_dim(size, 1, 1);
  calc_f<<< grid_dim, block_dim >>>(
    i_view,
    n_view,
    n_nonzero_items,
    interval,
    projected_centers,
    vol_tag,
    f_array );

  // check if kernel execution generated an error
  cutilCheckMsg("Kernel execution failed");


  // copy to sum_array for sum
  reduceSinglePass(n_nonzero_items, 256,
    (n_nonzero_items/256)+((n_nonzero_items%256)?1:0), 
    f_array, sum_array);

  // check if kernel execution generated an error
  cutilCheckMsg("Kernel execution failed");

  cutilSafeCall( hipDeviceSynchronize() );

  // copy and return result
  float result = 0.0f;
  cutilSafeCall( hipMemcpy(
    &result,
    sum_array,
    sizeof(float),
    hipMemcpyDeviceToHost ));

  cutilSafeCall( hipGetLastError() );

  return result;
}


void calculate_g_cuda( int      level, 
                       int      i_view, 
                       int      n_view,
                       int      n_nonzero_items,
                       int      interval,
                       uint16*  projected_centers, 
                       int*     vol_tag,
                       float*   g_array )
{
  int length = 1 << level;
  dim3 dim_grid(length, length, 1);
  dim3 dim_block(length, 1, 1);

  calc_g <<<dim_grid, dim_block>>> (
    i_view,
    n_view,
    n_nonzero_items,
    interval,
    projected_centers,
    vol_tag,
    g_array
    );

  // check if kernel execution generated an error
  cutilCheckMsg("Kernel execution failed");
  cutilSafeCall( hipGetLastError() );
}


////////////////////////////////////////
// read pptr volume to linear memory
////////////////////////////////////////
void get_volume_cuda( int level,
                      hipPitchedPtr vol_pptr,
                      //int * tag_vol,
                      float * den_vol )
{
  int length = 1 << level;
  dim3 dim_grid(length, length, 1);
  dim3 dim_block(length, 1, 1);

  get_volume<<<dim_grid, dim_block>>> (
    vol_pptr,
    //tag_vol,
    den_vol
    );
  // check if kernel execution generated an error
  cutilCheckMsg("Kernel execution failed");
  cutilSafeCall( hipGetLastError() );

  fprintf(stderr, "..Finished kernel call of \"get_volume\"\n");

}

/////////////////////////////