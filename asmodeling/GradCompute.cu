#include "hip/hip_runtime.h"
#include <cstdio>

#include <hip/hip_runtime.h>
#include <cutil_inline.h>
#include <cutil_math.h>

// parallel reduce kernel
#include "reduction_kernel.cu"

//////////////////////////////////////////////////////////////////////////////////////////////////////
//
//    GLOBAL VARIABLES
//
//////////////////////////////////////////////////////////////////////////////////////////////////////
texture<uchar4, 2, hipReadModeElementType> render_result;
texture<uchar4, 2, hipReadModeElementType> perturbed_result;

texture<uchar4, 3, hipReadModeElementType> ground_truth;


//////////////////////////////////////////////////////////////////////////////////////////////////////
//
//    Bind CUDA arrays to texture references
//
//////////////////////////////////////////////////////////////////////////////////////////////////////

// bind hipArray to texture reference
template<typename pixel_T, int dim>
void bind_tex(hipArray* data_array, texture<pixel_T, dim, hipReadModeElementType>& tex)
{
  // set texture parameters
  tex.normalized = false;                      // access with normalized texture coordinates
  tex.filterMode = hipFilterModePoint;        // linear interpolation
  tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
  if (dim>=2);
  tex.addressMode[1] = hipAddressModeWrap;
  if (dim>=3)
    tex.addressMode[2] = hipAddressModeWrap;
  if (dim>=4)
    tex.addressMode[3] = hipAddressModeWrap;

  // channel descriptor
  hipChannelFormatDesc channelDesc;
  cutilSafeCall( hipGetChannelDesc(&channelDesc, data_array) );

  // bind array to 3D texture
  cutilSafeCall(hipBindTextureToArray(tex, data_array, channelDesc));
}

// bind input array to render result tex ref
void bind_rrtex_cuda(hipArray* data_array)
{
  bind_tex<uchar4,2>(data_array, render_result);
}

// bind input array to perturbed result tex ref
void bind_prtex_cuda(hipArray* data_array)
{
  bind_tex<uchar4,2>(data_array, perturbed_result);
}

// bind input array to ground truth tex ref
void bind_gttex_cuda(hipArray* data_array)
{
  bind_tex<uchar4,3>(data_array, ground_truth);
}


// map (i,j,k) to 1D array index
__device__ int index3(int i, int j, int k, int length)
{
  return i + length * ( j + k * length );
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//
//  Reconstruct the volume on the cuda
//
/////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void construct_volume(
                                 hipPitchedPtr vol_pptr, 
                                 float * device_x, 
                                 int * tag_vol
                                 )
{
  //unsigned int z = blockIdx.x;
  //unsigned int y = blockIdx.y;
  //unsigned int x = threadIdx.x;
  // slice_pitch = pitch * height
  // slice = ptr + z * slice_pitch
  // row = (type*)(slice + y*pitch)
  // elem = *(row + x)

  int index = index3(threadIdx.x, blockIdx.y, blockIdx.x, blockDim.x);

  char * slice = (char *)vol_pptr.ptr + blockIdx.x * vol_pptr.pitch * blockDim.x;

  *((float*)(slice+blockIdx.y*vol_pptr.pitch) + threadIdx.x)
    = device_x[ tag_vol[index] ];
}
void construct_volume_cuda(
                           float * device_x,
                           hipPitchedPtr * density_vol,
                           hipExtent extent,
                           int *   tag_vol
                           )
{
  dim3 grid_dim(extent.depth, extent.height, 1);
  dim3 block_dim(extent.width/4, 1, 1);

  construct_volume<<< grid_dim, block_dim >>>(
    *density_vol,
    device_x,
    tag_vol
    );
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//
//  Upsample current level to max_level
//
/////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void upsample_volume(
                                hipPitchedPtr pptr_lower,
                                hipExtent     extent_lower,
                                hipPitchedPtr pptr_higher,
                                int scale
                                )
{
  // for higher level volume
  unsigned int k = blockIdx.x;
  unsigned int j = blockIdx.y;
  unsigned int i = threadIdx.x;

  char * slice = (char*)pptr_higher.ptr + k*pptr_higher.pitch*blockDim.x;
  float *p_higher = (float*)(slice + j*pptr_higher.pitch) + i;

  k /= scale;
  j /= scale;
  k /= scale;

  slice = (char*)pptr_lower.ptr + k*pptr_lower.pitch*extent_lower.depth;
  float *p_lower  = (float*)(slice + j*pptr_lower.pitch) + i;

  *p_higher = *p_lower;
}
void upsample_volume_cuda(
                          int level,
                          int max_level,
                          hipPitchedPtr * lower_lev,
                          hipPitchedPtr * upper_lev
                          )
{
  int length = 1 << level;
  int max_length = 1 << max_level;

  dim3 grid_dim(max_length, max_length, 1);
  dim3 block_dim(max_length, 1, 1);

  hipExtent ext_low = make_hipExtent(length, length, length);
  int scale = max_length / length;

  upsample_volume<<< grid_dim, block_dim >>>(
    *lower_lev,
    ext_low,
    *upper_lev,
    scale );
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//
//  Calc F on the cuda
//
/////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void calc_f(
                       int i_view,           // which view, relates to ofset in proj_centers...
                       int n_view,           // num of different views/cameras
                       int n,                // num of items in array x, f, and g
                       int interval,         // the occupycation radius of projection
                       int * proj_centers,   // 
                       int * tag_vol,
                       float * f_array
                       )
{
  unsigned int i = threadIdx.x;
  unsigned int j = blockIdx.y;
  unsigned int k = blockIdx.x;

  int index_vol   = index3(i, j, k, blockDim.x);
  int index_array = tag_vol[index_vol];

  if (index_array != 0)
  {
    // pixel on the image
    int u = proj_centers[n_view * 2 * (index_array-1)];
    int v = proj_centers[n_view * 2 * (index_array-1)+1];

    float f = 0.0;
    for (int uu = u - interval; uu <= u + interval; ++u)
    {
      for (int vv = v - interval; vv <= v + interval; ++v)
      {
        uchar4 rr4 = tex2D(render_result, float(uu), float(vv));
        uchar4 gt4 = tex3D(ground_truth, float(uu), float(vv), float(i_view));
        // USE ONLY R CHANNEL HERE...
        f += (rr4.x-gt4.x)*(rr4.x-gt4.x)/(255.0*255.0);
      }
    }
    f_array[ index_array ] = f;
  } // if (index_array != 0)
}

extern "C"
void reduceSinglePass(int size, int threads, int blocks, float *d_idata, float *d_odata);
float calculate_f_cuda(
                       int    level, 
                       int    i_view, 
                       int    n_view,
                       int    n_nonzero_items,
                       int    interval,
                       int*   projected_centers, 
                       int*   vol_tag,
                       float* f_array,
                       float* sum_array
                       )
{
  int size = 1 << level;

  // calc f value for each non-zero cell
  dim3 grid_dim(size, size, 1);
  dim3 block_dim(size, 1, 1);
  calc_f<<< grid_dim, block_dim >>>(
    i_view,
    n_view,
    n_nonzero_items,
    interval,
    projected_centers,
    vol_tag,
    f_array );

  cutilSafeCall( hipDeviceSynchronize() );

  // copy to sum_array for sum
  reduceSinglePass(n_nonzero_items, 256,
    (n_nonzero_items/256)+((n_nonzero_items%256)?1:0), 
    f_array, sum_array);

  // copy and return result
  float result;
  hipMemcpy(
    &result,
    sum_array,
    sizeof(float),
    hipMemcpyDeviceToHost);
  return result;
}
////////////////////////////////////////////////////////////////////////////////////////////////////
//
//  Calc G[] on the cuda
//
/////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void calc_g()
{
}

void calculate_g_cuda(int level, int max_level, int* projected_centers, float* g_array)
{
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//
//  set parameters on CUDA
//
/////////////////////////////////////////////////////////////////////////////////////////////////////
